// v3
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <fstream>

using namespace std;
using namespace std::chrono;

#define CHECK_CUDA_ERROR(err) \
    { \
        if (err != hipSuccess) { \
            cerr << "CUDA Error: " << hipGetErrorString(err) << endl; \
            exit(1); \
        } \
    }

__global__ void gameOfLifeKernel(int *current, int *next, int boardSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < boardSize && col < boardSize) {
        int aliveNeighbors = 0;
        for (int i = -1; i <= 1; ++i) {
            for (int j = -1; j <= 1; ++j) {
                if (i == 0 && j == 0) continue;
                int newRow = row + i;
                int newCol = col + j;
                if (newRow >= 0 && newRow < boardSize && newCol >= 0 && newCol < boardSize) {
                    aliveNeighbors += current[newRow * boardSize + newCol];
                }
            }
        }
        int index = row * boardSize + col;
        next[index] = (current[index] == 1) ? (aliveNeighbors < 2 || aliveNeighbors > 3 ? 0 : 1) : (aliveNeighbors == 3 ? 1 : 0);
    }
}

void initializeBoard(int *board, int boardSize) {
    srand(12345); 
    for (int i = 0; i < boardSize * boardSize; ++i) {
        board[i] = rand() % 2;
    }
}

// final board to file
void writeFinalBoardToFile(const int *board, int n, int iterations, const string &outputDir)
{
    string correctedOutputDir = outputDir;
    if (outputDir.back() != '/')
    {
        correctedOutputDir += "/";
    }

    string fileName = correctedOutputDir + "hw5_GPU_" + to_string(n) + "x" + to_string(n) +
                      "_board_" + to_string(iterations) + "_iterations_V3code_OptimizedV2_testcase.txt";

    ofstream outFile(fileName);

    if (!outFile)
    {
        printf("Error creating output file: %s\n", fileName.c_str());
        return;
    }

    for (int i = 0; i < n; ++i)
    { 
        for (int j = 0; j < n; ++j)
        {
            outFile << (board[i * n + j] ? '*' : '.') << " ";
        }
        outFile << endl;
    }

    outFile.close();
    printf("Final board written to %s\n", fileName.c_str());
}

int main(int argc, char *argv[]) {
    if (argc != 4) { 
        cout << "Usage: " << argv[0] << " <board size> <generations> <output directory>" << endl;
        return 1;
    }

    int boardSize = stoi(argv[1]);
    int generations = stoi(argv[2]);
    string outputDir = argv[3];

    size_t size = boardSize * boardSize * sizeof(int);
    int *h_current = new int[boardSize * boardSize];
    int *h_next = new int[boardSize * boardSize];

    initializeBoard(h_current, boardSize);

    int *d_current, *d_next;
    CHECK_CUDA_ERROR(hipMalloc(&d_current, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_next, size));
    CHECK_CUDA_ERROR(hipMemcpy(d_current, h_current, size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((boardSize + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (boardSize + threadsPerBlock.y - 1) / threadsPerBlock.y);

    auto start = high_resolution_clock::now();

    for (int gen = 0; gen < generations; ++gen) {
        gameOfLifeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_current, d_next, boardSize);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipMemcpy(d_current, d_next, size, hipMemcpyDeviceToDevice));
    }

    auto end = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(end - start);

    CHECK_CUDA_ERROR(hipMemcpy(h_next, d_current, size, hipMemcpyDeviceToHost));

    writeFinalBoardToFile(h_next, boardSize, generations, outputDir);

    cout << "Simulation completed in " << duration.count() << " ms." << endl;

    hipFree(d_current);
    hipFree(d_next);
    delete[] h_current;
    delete[] h_next;

    return 0;
}
