#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <fstream>
using namespace std;
using namespace std::chrono;
#define CHECK_CUDA_ERROR(err) \
    { \
        if (err != hipSuccess) { \
            cerr << "CUDA Error: " << hipGetErrorString(err) << endl; \
            exit(1); \
        } \
    }
__global__ void gameOfLifeKernel(const int *__restrict__ current, int *next, int boardSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < boardSize && col < boardSize) {
        int index = row * boardSize + col;
        int up = row - 1;
        int down = row + 1;
        int left = col - 1;
        int right = col + 1;
        int aliveNeighbors = 0;
        if (up >= 0 && left >= 0)
            aliveNeighbors += __ldg(&current[up * boardSize + left]);
        if (up >= 0)
            aliveNeighbors += __ldg(&current[up * boardSize + col]);
        if (up >= 0 && right < boardSize)
            aliveNeighbors += __ldg(&current[up * boardSize + right]);
        if (left >= 0)
            aliveNeighbors += __ldg(&current[row * boardSize + left]);
        if (right < boardSize)
            aliveNeighbors += __ldg(&current[row * boardSize + right]);
        if (down < boardSize && left >= 0)
            aliveNeighbors += __ldg(&current[down * boardSize + left]);
        if (down < boardSize)
            aliveNeighbors += __ldg(&current[down * boardSize + col]);
        if (down < boardSize && right < boardSize)
            aliveNeighbors += __ldg(&current[down * boardSize + right]);
        int cellState = __ldg(&current[index]);
        next[index] = (cellState == 1) ? (aliveNeighbors < 2 || aliveNeighbors > 3 ? 0 : 1) : (aliveNeighbors == 3 ? 1 : 0);
    }
}
void initializeBoard(int *board, int boardSize) {
    srand(12345);
    for (int i = 0; i < boardSize * boardSize; ++i) {
        board[i] = rand() % 2;
    }
}
void writeFinalBoardToFile(const int *board, int n, int iterations, const string &outputDir)
{
    string correctedOutputDir = outputDir;
    if (outputDir.back() != '/')
    {
        correctedOutputDir += "/";
    }
    string fileName = correctedOutputDir + "hw5_GPU_" + to_string(n) + "x" + to_string(n) +
                      "_board_" + to_string(iterations) + "_iterations_V3code_Optimized_testcase.txt";
    ofstream outFile(fileName);
    if (!outFile)
    {
        printf("Error creating output file: %s\n", fileName.c_str());
        return;
    }
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            outFile << (board[i * n + j] ? '*' : '.') << " ";
        }
        outFile << endl;
    }
    outFile.close();
    printf("Final board written to %s\n", fileName.c_str());
}
int main(int argc, char *argv[]) {
    if (argc != 4) {
        cout << "Usage: " << argv[0] << " <board size> <generations> <output directory>" << endl;
        return 1;
    }
    int boardSize = stoi(argv[1]);
    int generations = stoi(argv[2]);
    string outputDir = argv[3];
    size_t size = boardSize * boardSize * sizeof(int);
    int *h_current = new int[boardSize * boardSize];
    int *h_next = new int[boardSize * boardSize];
    initializeBoard(h_current, boardSize);
    int *d_current, *d_next;
    CHECK_CUDA_ERROR(hipMalloc(&d_current, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_next, size));
    CHECK_CUDA_ERROR(hipMemcpy(d_current, h_current, size, hipMemcpyHostToDevice));
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((boardSize + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (boardSize + threadsPerBlock.y - 1) / threadsPerBlock.y);
    auto start = high_resolution_clock::now();
    for (int gen = 0; gen < generations; ++gen) {
        gameOfLifeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_current, d_next, boardSize);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipMemcpy(d_current, d_next, size, hipMemcpyDeviceToDevice));
    }
    auto end = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(end - start);
    CHECK_CUDA_ERROR(hipMemcpy(h_next, d_current, size, hipMemcpyDeviceToHost));
    writeFinalBoardToFile(h_next, boardSize, generations, outputDir);
    cout << "Simulation completed in " << duration.count() << " ms." << endl;
    hipFree(d_current);
    hipFree(d_next);
    delete[] h_current;
    delete[] h_next;
    return 0;
}
