// game of life for HW5 GPU VERSION

// going to test code on cluster as I don't have NVDIA GPU but starting with a simple addition until number 10 loop

// Instructions to compile the program: nvcc -o gameoflife gameoflife.cu

// Instructions to run the program: ./gameoflife

#include <iostream>
#include <hip/hip_runtime.h>

// kernel function to count to 10
__global__ void count_to_ten() {
    int thread_id = threadIdx.x; // get the thread index

    // only let the first thread in the block count to 10
    if (thread_id == 0) {
        for (int i = 1; i <= 10; i++) {
            printf("Count: %d\n", i);
        }
    }
}

int main() {
    std::cout << "Starting the GPU count to 10 program..." << std::endl;

    // launch the kernel with 1 block and 1 thread
    count_to_ten<<<1, 1>>>();

    // synchronize GPU and CPU to ensure all output is printed
    hipDeviceSynchronize();

    std::cout << "Finished counting on GPU." << std::endl;

    return 0;
}
